#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                  \
    {                                                                     \
        hipError_t err = call;                                           \
        if (err != hipSuccess)                                           \
        {                                                                 \
            fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err)); \
            return -1;                                                    \
        }                                                                 \
    }

__global__ void normalize(hipfftComplex *data, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        data[idx].x /= N; // normalize real part
        data[idx].y /= N; // normalize imaginary part
    }
}

extern "C" void *unified_malloc(size_t size)
{
    void *ptr;
    hipError_t err = hipMallocManaged(&ptr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        // lmao totally fine..don't worry.
        return NULL;
    }
    return ptr;
}

extern "C" size_t unified_free(void *ptr)
{
    CUDA_CHECK(hipFree(ptr));
    return 0;
}

extern "C" size_t perform_cuda_unified(float *buffer, size_t buffer_size, hipfftComplex *output)
{
    const int BATCH = 16; // arbitrarily chosen
    hipfftHandle plan;
    if (hipfftPlan1d(&plan, buffer_size, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT Error: Plan creation failed\n");
        return EXIT_FAILURE;
    }

    if (hipfftExecR2C(plan, buffer, output) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT Error: ExecR2C failed\n");
        return EXIT_FAILURE;
    }

    int blockSize = 256;
    int total_output = (buffer_size / 2 + 1) * BATCH;
    int numBlocks = (total_output + blockSize - 1) / blockSize;
    normalize<<<numBlocks, blockSize>>>(output, total_output);

    CUDA_CHECK(hipDeviceSynchronize());

    hipfftDestroy(plan);

    return 0;
}
